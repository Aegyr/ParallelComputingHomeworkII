#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include "cuda_utils.h"

__constant__ int TILE_WIDTH;

__global__ void MatrixMulKernel(float* Md, float* Nd, float* Pd, int Width)
{
	// Calculate the row index of the Pd element and M
	int Row = blockIdx.y*TILE_WIDTH + threadIdx.y;
	// Calculate the column index of Pd and N
	int Col = blockIdx.x*TILE_WIDTH + threadIdx.x;
	float Pvalue = 0;
	// each thread computes one element of the block sub-matrix
	
	for (int k = 0; k < Width; ++k)
	{
		// printf("\n Row/Col %d/%d Pvalue += A[%d] * B[%d]",Row,Col,Row*Width+k,k*Width+Col);
		Pvalue += Md[Row*Width+k] * Nd[k+Width*Col];
	}
	Pd[Row*Width+Col] = Pvalue;
}

int main( int argc, char* argv[] )
{
	Timer timer ;
	double duration;	
	
	int ToDo = 2;
	
	float *matrixA_h, *matrixB_h, *matrixA_d, *matrixB_d, *matrix_erg, *matrix_erg_d;
		
	int dimensionAx = 2;
	int dimensionAy = 2;
	int dimensionBy = 2;
	int dimensionBx = 2;
	
	int sizeA = dimensionAx * dimensionAy * sizeof(float);
	int sizeB = dimensionBx * dimensionBy * sizeof(float);
	
	matrixA_h  = (float*) malloc(sizeA);
	matrixB_h  = (float*) malloc(sizeB);
	matrix_erg = (float*) malloc(dimensionAy * dimensionBx * sizeof(float)); 
	
	int dimG = 2;				
	int dimBX = (dimensionAx/dimG);
	int dimBY = (dimensionBy/dimG);

	hipMemcpyToSymbol(HIP_SYMBOL(&TILE_WIDTH),&dimBX,sizeof(int),0,hipMemcpyHostToDevice);
	
	dim3 gridDim(dimG,dimG),blockDim(dimBX,dimBY);
	
	int x,y;
	/*
	 Initialize A and B^T like:
	 |111....	|
	 |222....	|
	 |333		|
	 |.			|
	 |.			|
	 A is stored row wise
	 B is stores column wise 
	 erg is stored row wise
	*/
	for(y=0; y<dimensionAy; y++)
	{
		for(x=0; x<dimensionAx; x++)
		{
			matrixA_h[x+(dimensionAx*y)] = y+1;
		}
	}

	for(x=0; x<dimensionBx; x++)
	{
		for(y=0; y<dimensionBy; y++)
		{
			matrixB_h[y+(dimensionBy*x)] = y+1;
		}
	}
	/* // Print the Matrix
	printf("\nA:\n");
	for(x=0 ; x < dimensionAx*dimensionAy; x++)
	{
		printf(" %f", matrixA_h[x]);
	}
	printf("\nB:\n");
	for(x=0 ; x < dimensionBx*dimensionBy; x++)
	{
		printf(" %f", matrixB_h[x]);
	}
	
	printf("\n nach dem initialisieren \n");
	*/
	switch(ToDo)
	{
		case 1:
			initTimer (& timer );
			int row,column,columnB;
			/*********************************
 			 *	Matrix Multiplikation on CPU * 
			 *********************************/
			for(row=0; row<dimensionAy; row++)
			{
				for(column=0; column<dimensionBx; column++)
				{
					//printf("\nErg[%d]=",(row*dimensionBx)+column);
					matrix_erg[(row*dimensionBx)+column] = 0;						
					for(columnB=0; columnB<dimensionBx; columnB++)
					{
						//printf("+A[%d]*B[%d]", columnB+(dimensionAx*row),columnB+(dimensionBy*column));
						matrix_erg[(row*dimensionBx)+column] += matrixA_h[columnB+(dimensionAx*row)] *matrixB_h[columnB+(dimensionBy*column)];
					}
				}
			}
			for(x=0; x<dimensionAy*dimensionBy; x++)
			{
				printf("\n erg[%d] %f ", x, matrix_erg[x]);
			}
//			cudaThreadSyncronize(); // identifier undefined ???
			duration = getTimer(&timer);
		break;
		case 2: 
			/* Matrix Multiplikation on GPU with device Mem*/
			/****************************************************
 			 *	Matrix Multiplikation on GPU without shared Mem * 
			 ****************************************************/
			 // ToDo: solution for non squared matrix
			 // 
				hipMalloc(&matrixA_d, sizeA);
				hipMalloc(&matrixB_d, sizeB);
				hipMalloc(&matrix_erg_d,dimensionAx*dimensionBy*sizeof(float));
				hipMemcpy(matrixA_d,matrixA_h,sizeA, hipMemcpyHostToDevice);
				hipMemcpy(matrixB_d,matrixB_h,sizeB, hipMemcpyHostToDevice);
				
				/* need: 
				 *	maximal thread per block number
				 *  maximal number of block which run at the same time
				 *
				*/
				// {dimensionX | dimensionX € N , dimensionX % 2 = 0} 
				// {dimensionY | dimensionY € N , dimensionY % 2 = 0} 

				MatrixMulKernel<<<gridDim,blockDim>>>(matrixA_d,matrixB_d,matrix_erg_d,dimensionAx);

				hipMemcpy(matrix_erg, matrix_erg_d,dimensionAx*dimensionBy*sizeof(float) , hipMemcpyDeviceToHost);
				for(x=0; x<dimensionAy*dimensionBy; x++)
				{
					printf("\n erg[%d] %f ", x, matrix_erg[x]);
				}
		break;
		case 3: 
			/* Matrix Multiplikation on GPU with shared Mem*/
			//TODO

		
		break;

	}
	
	hipFree(matrixA_d);
	hipFree(matrixB_d);
	hipFree(matrix_erg_d);
	free(matrixA_h);
	free(matrixB_h);
	free(matrix_erg);
	
	return 0;
}
