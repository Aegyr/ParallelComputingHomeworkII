#include "hip/hip_runtime.h"
#include "treeOperations.h"

#define SUBDIV 2
#define THREADS 128

__global__ void testCase(int nx, int ny, int* res)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	TreeElem* gpuTree;
	createFullTree(&gpuTree, nx, ny, SUBDIV);
	hipDeviceSynchronize();
	res[i] = countLeaves(gpuTree);
	delete gpuTree;
}

int main(int argc, char** argv)
{
	int nx = 16;
	int ny = 16;

	if (argc > 1)
	{
		nx = (int)atoi(argv[1]);
		ny = nx;
	}
	else if (argc > 2)
	{
		nx = (int)atoi(argv[1]);
		ny = (int)atoi(argv[2]);
	}

	hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, 5);
	hipDeviceSetLimit(cudaLimitDevRuntimePendingLaunchCount, 32768);

	size_t lim;
	hipDeviceGetLimit(&lim, cudaLimitDevRuntimeSyncDepth);
	std::cout << "rsd: " << lim << std::endl;

	lim;
	hipDeviceGetLimit(&lim, cudaLimitDevRuntimePendingLaunchCount);
	std::cout << "plc: " << lim << std::endl;

	std::cout << "nx: " << nx << " ny: " << ny << std::endl;
	std::cout << "numThreadsX: " << THREADS << " numThreadsY: " << THREADS << std::endl;

	int res[THREADS * THREADS];
	int* res_d;
	checkCudaErrors(hipMalloc(&res_d, THREADS * THREADS * sizeof(int)));

	dim3 block(16 * 16);
	dim3 grid(THREADS * THREADS / (16 * 16));
	testCase << <grid, block>> >(nx, ny, res_d);
	checkCudaErrors(hipGetLastError());

	checkCudaErrors(hipMemcpy(res, res_d, THREADS * THREADS * sizeof(int), hipMemcpyDeviceToHost));

	for (int i = 0; i < THREADS * THREADS; i++)
		std::cout << res[i] << std::endl;

	checkCudaErrors(hipFree(res_d));

	hipDeviceReset();
	return 0;
}
