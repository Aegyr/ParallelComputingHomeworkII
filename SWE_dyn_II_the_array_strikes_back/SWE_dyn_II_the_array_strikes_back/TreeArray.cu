#include "TreeArray.h"

TreeArray::TreeArray(int x, int y)
{
	nx = x;
	ny = y;

	hipMallocManaged(&values, nx * ny * sizeof(float));
	hipMallocManaged(&depths, nx * ny * sizeof(int));
}

TreeArray::TreeArray(const TreeArray& other)
{
	nx = other.nx;
	ny = other.ny;

	hipMallocManaged(&values, nx * ny * sizeof(float));
	hipMallocManaged(&depths, nx * ny * sizeof(float));

	memcpy(values, other.values, nx * ny * sizeof(float));
	memcpy(depths, other.depths, nx * ny * sizeof(float));
}

TreeArray::~TreeArray()
{
	//crashes???
	//hipFree(values);
	//hipFree(depths);
}